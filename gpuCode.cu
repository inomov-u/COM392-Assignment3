#include "hip/hip_runtime.h"
/*******************************************************************************
*
*   COMMENTS GO HERE
*
*   TODO LIST GOES HERE
*
*******************************************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpuCode.h"
#include "params.h"

/******************************************************************************/
// return information about CUDA GPU devices on this machine
int probeGPU(){

  hipError_t err;
  err = hipDeviceReset();

  hipDeviceProp_t prop;
  int count;
  err = hipGetDeviceCount(&count);
  if(err != hipSuccess){
    printf("problem getting device count = %s\n", hipGetErrorString(err));
    return 1;
    }
  printf("number of GPU devices: %d\n\n", count);

  for (int i = 0; i< count; i++){
    printf("************ GPU Device: %d ************\n\n", i);
    err = hipGetDeviceProperties(&prop, i);
    if(err != hipSuccess){
      printf("problem getting device properties = %s\n", hipGetErrorString(err));
      return 1;
      }

    printf("\tName: %s\n", prop.name);
    printf( "\tCompute capability: %d.%d\n", prop.major, prop.minor);
    printf( "\tClock rate: %d\n", prop.clockRate );
    printf( "\tDevice copy overlap: " );
      if (prop.deviceOverlap)
        printf( "Enabled\n" );
      else
        printf( "Disabled\n" );
    printf( "\tKernel execition timeout: " );
      if (prop.kernelExecTimeoutEnabled)
        printf( "Enabled\n" );
      else
        printf( "Disabled\n" );
    printf( "--- Memory Information for device %d ---\n", i );
    printf("\tTotal global mem: %ld\n", prop.totalGlobalMem );
    printf("\tTotal constant Mem: %ld\n", prop.totalConstMem );
    printf("\tMax mem pitch: %ld\n", prop.memPitch );
    printf( "\tTexture Alignment: %ld\n", prop.textureAlignment );
    printf("\n");
    printf( "\tMultiprocessor count: %d\n", prop.multiProcessorCount );
    printf( "\tShared mem per processor: %ld\n", prop.sharedMemPerBlock );
    printf( "\tRegisters per processor: %d\n", prop.regsPerBlock );
    printf( "\tThreads in warp: %d\n", prop.warpSize );
    printf( "\tMax threads per block: %d\n", prop.maxThreadsPerBlock );
    printf( "\tMax block dimensions: (%d, %d, %d)\n",
                  prop.maxThreadsDim[0],
                  prop.maxThreadsDim[1],
                  prop.maxThreadsDim[2]);
    printf( "\tMax grid dimensions: (%d, %d, %d)\n",
                  prop.maxGridSize[0],
                  prop.maxGridSize[1],
                  prop.maxGridSize[2]);
    printf("\n");
  }

return 0;
}

/******************************************************************************/
int updatePalette(GPU_Palette* P){

  updateReds <<< P->gBlocks, P->gThreads >>> (P->red);
  //updateGreens <<< P->gBlocks, P->gThreads >>> (P->green);
	updateBlues <<< P->gBlocks, P->gThreads >>> (P->blue);

  runRotation <<< P->gBlocks, P->gThreads >>> (P->green, P->red, P->blue, P->num_pixels);

  return 0;
}

/******************************************************************************/
__global__ void updateReds(float* red){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  red[vecIdx] = 0;

}

/******************************************************************************/
__global__ void runRotation(float* green, float* red, float* blue, long strandLength){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  //blue[tid] = green[strandLength - 1 - tid];
  // for(long i = startIdx; i < endIdx; i++){  // i = start index of strand2
  float acc = 0.0;
  long the_index;
  for(long j = 0; j < strandLength; j++){
    the_index = j+tid;
    if (the_index >= strandLength) the_index = the_index - strandLength;
    acc += abs((green[j] - green[strandLength - 1 - the_index]));
  }

  float avg = (float) (acc * 1.0 )/strandLength; // score for the rotation


  red[tid] = avg;
}

/******************************************************************************/
__global__ void updateGreens(float* green){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  green[vecIdx] = 0;
}

/******************************************************************************/
__global__ void updateBlues(float* blue){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  blue[vecIdx] = 0;
}


/******************************************************************************/
GPU_Palette initGPUPalette(unsigned int imageWidth, unsigned int imageHeight)
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = ceil(imageWidth/32);  // however many blocks needed for image
  X.gBlocks.y = ceil(imageHeight/32);
  X.gBlocks.z = 1;

  X.palette_width = imageWidth;       // save this info
  X.palette_height = imageHeight;
  X.num_pixels = imageWidth * imageHeight;
  X.memSize =  imageWidth * imageHeight * sizeof(float);

  // allocate memory on GPU
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.green, X.memSize); // g
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.blue, X.memSize);  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

  return X;
}

/******************************************************************************/
int freeGPUPalette(GPU_Palette* P) {

  // free gpu memory
  hipFree(P->gray);
  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->blue);

  return 0;
}

/*************************************************************************/
